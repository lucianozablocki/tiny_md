#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "core.h"  // Shared header
#include "parameters.h"

#define ECUT (4.0f * (powf(RCUT, -12.0f) - powf(RCUT, -6.0f)))

__device__ float3 minimum_image(float3 rij, float L) {
    rij.x -= rintf(rij.x / L) * L;
    rij.y -= rintf(rij.y / L) * L;
    rij.z -= rintf(rij.z / L) * L;
    return rij;
}

__global__ void forces_kernel(const float4* rxyz, float4* fxyz, 
                             float* epot, float* pres_vir,
                             float rcut2, float L) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    float4 ri = rxyz[i];
    float4 fi = {0.0f, 0.0f, 0.0f, 0.0f};
    float local_epot = 0.0f;
    float local_pres_vir = 0.0f;

    for (int j = i+1; j < N; j++) {

        float4 rj = rxyz[j];
        float3 rij = {ri.x - rj.x, ri.y - rj.y, ri.z - rj.z};
        rij = minimum_image(rij, L);

        float rij2 = rij.x*rij.x + rij.y*rij.y + rij.z*rij.z;
        if (rij2 <= rcut2) {
            float r2inv = 1.0f / rij2;
            float r6inv = r2inv * r2inv * r2inv;
            float fr = 24.0f * r2inv * r6inv * (2.0f * r6inv - 1.0f);

            fi.x += fr * rij.x;
            fi.y += fr * rij.y;
            fi.z += fr * rij.z;

			// Update force on j (REQUIRES ATOMIC)
            atomicAdd(&fxyz[j].x, -fr * rij.x);
            atomicAdd(&fxyz[j].y, -fr * rij.y);
            atomicAdd(&fxyz[j].z, -fr * rij.z);

            local_epot += 4.0f * r6inv * (r6inv - 1.0f) - ECUT;
            local_pres_vir += fr * rij2;
        }
    }

	atomicAdd(&fxyz[i].x, fi.x);
	atomicAdd(&fxyz[i].y, fi.y);
	atomicAdd(&fxyz[i].z, fi.z);
    atomicAdd(epot, local_epot);
    atomicAdd(pres_vir, local_pres_vir);
}

void forces(const float* rxyz, float* fxyz, float* epot, float* pres,
                const float* temp, float rho, float V, float L) {
    float *d_rxyz, *d_fxyz, *d_epot, *d_pres_vir;
    size_t size = 4 * N * sizeof(float);

    hipMalloc(&d_rxyz, size);
    hipMalloc(&d_fxyz, size);
    hipMalloc(&d_epot, sizeof(float));
    hipMalloc(&d_pres_vir, sizeof(float));

    hipMemcpy(d_rxyz, rxyz, size, hipMemcpyHostToDevice);
    hipMemset(d_fxyz, 0, size);
    hipMemset(d_epot, 0, sizeof(float));
    hipMemset(d_pres_vir, 0, sizeof(float));

    dim3 blocks((N + 127) / 128);
    dim3 threads(128);
    forces_kernel<<<blocks, threads>>>((float4*)d_rxyz, (float4*)d_fxyz, 
                                     d_epot, d_pres_vir, RCUT*RCUT, L);

    hipMemcpy(fxyz, d_fxyz, size, hipMemcpyDeviceToHost);
    hipMemcpy(epot, d_epot, sizeof(float), hipMemcpyDeviceToHost);
    
    float h_pres_vir;
    hipMemcpy(&h_pres_vir, d_pres_vir, sizeof(float), hipMemcpyDeviceToHost);
    *pres = *temp * rho + h_pres_vir / (3.0f * V);

    hipFree(d_rxyz);
    hipFree(d_fxyz);
    hipFree(d_epot);
    hipFree(d_pres_vir);
}
